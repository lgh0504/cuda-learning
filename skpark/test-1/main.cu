#include "hip/hip_runtime.h"
#include <thrust/version.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include <ctime>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand.h>

using namespace std;

__global__ void vectoradd(float* a,float* b, float* c){
    unsigned int thread_id=threadIdx.x + blockIdx.x*blockDim.x;
    c[thread_id]=a[thread_id]+b[thread_id];
}

int main(){
    clock_t timestamp=clock();
    hipFree(0);

    int N=1000;
    size_t size=N*sizeof(float);

    float* h_a=(float*)malloc(size);    
    float* h_b=(float*)malloc(size);
    float* h_c=(float*)malloc(size);

    for(int i=0;i<N;i++)
    {
        h_a[i]=1.0f;
        h_b[i]=3.0f;
        h_c[i]=0.0f;
    }

    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,size,hipMemcpyHostToDevice);    
    vectoradd<<<4,256>>>(d_a,d_b,d_c);
    hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++)
        cout << h_c[i] << " ";
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);
    cout << "hekll"<<endl << timestamp <<endl;
    return 0;
}